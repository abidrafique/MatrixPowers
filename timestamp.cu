

#include <hip/hip_runtime.h>
#include <sys/time.h>


double timestamp ()
{
	struct timeval tv;
	gettimeofday (&tv, 0);
	return tv.tv_sec + 1e-6*tv.tv_usec;
}
