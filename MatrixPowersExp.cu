#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>

#include <cusp/io/matrix_market.h>
#include <cusp/dia_matrix.h>
#include <cusp/array2d.h>
#include <cusp/array1d.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include<stdio.h>
#include<iostream>
#include"timestamp.h"
#include"cuda_macros.h"
#include"matrix_powers_kernel.h"
#include"Print.h"
#define DISPLAY        0 


using namespace std;
int main(int argc, char* argv[])
{
   
   int S_PARAM1= S_PARAM_LIMIT;
//   int Matrices[NUM_MATRICES] ={2000,5000,20000,50000,120000,250000,500000,1000000}; 
   int MatrixSize  = 0;
   float gflops3  = 0;

   //for( int i =0; i<8;i++){//NUM_MATRICES;i++)
{
   
//    MatrixSize = Matrices[i];
    MatrixSize = atoi(argv[1]);

   const int ROWS = MatrixSize;
   int Errors =0;

   char filename[200];
   //sprintf(filename,"/home/abid/Tutorials/MatrixPowers/Benchmarks/band%d_n_%d.mtx",BAND_SIZE,MatrixSize);	
//   sprintf(filename,"Benchmarks/band%d_n_%d.mtx",BAND_SIZE,MatrixSize);	
   sprintf(filename,"%s",argv[2]);	
 
   cusp::dia_matrix<int, float, cusp::device_memory> A;
   cusp::io::read_matrix_market_file(A, filename);
   int n = A.num_cols;
   int m = A.num_rows;
   int num_entries = A.num_entries;
	   
   cusp::dia_matrix<int, float, cusp::host_memory> A_host=A;


 //Generaating Random x	
    cusp::array1d<float, cusp::host_memory> x_host(n);	
//    for(int k=0;k<n;k++)
//	x_host[k] = rand()%10;
   float sum=0; 
    for(int k=0;k<n;k++)
    {
//	x_host[k] = k;//rand()%10;
	sum += (float)k*(float)k; 
	}
    float normx = sqrt(sum);
    for(int k=0;k<n;k++)
    {
	x_host[k] = (float)k/normx;//rand()%10;
//	sum += k*k; 
	}
    
    cusp::array1d<float, cusp::device_memory> x=x_host;	
    cusp::array1d<float, cusp::device_memory> y(m, 0);	

    cuda_sync();
    double ts = timestamp();
    for(int k=0;k<S_PARAM1;k++){
	if(k%2 == 0)
	    cusp::multiply(A,x,y);
	else
	    cusp::multiply(A,y,x);
    }
    cuda_sync();
    double total = timestamp() - ts;
    double flops = S_PARAM1*(2.0*(double) num_entries) - 1;
    double gflops = flops / 1000000000.0;
    gflops = gflops / total;
    if(DISPLAY)
	    cout << "SpMV                           " << total/flops << " seconds. " << gflops << " GFLOPS/s" <<endl;
    else
       	cout << m<<"\t"<<S_PARAM1<<"\t"<< gflops <<endl; 
    cusp::array1d<float, cusp::host_memory> y_host =x ;	
    if(S_PARAM1%2 == 1)	 
	    y_host =y ;	
    for(int S_PARAM=S_PARAM_LIMIT;S_PARAM>=1;S_PARAM--)
   // int S_PARAM=S_PARAM_LIMIT;
    {
	
	float* A_ptr = thrust::raw_pointer_cast(&A_host.values.values[0]);

	float* d_a;
	float* d_x;
	float* d_y;

	float* a = new float[ROWS*COLS];
	float* x1 = new float[ROWS];
	float* y1 = new float[S_PARAM*ROWS];
	for(int i=0;i<COLS;i++){
		for(int j=0;j<ROWS;j++){
			a[i*ROWS+j] =A_ptr[i*ROWS+j];
		}
	}
	for(int i=0;i<ROWS;i++){
		x1[i] = x_host[i];
	}
	hipMalloc(&d_a,ROWS*COLS*sizeof(float));	
	hipMalloc(&d_x,(ROWS)*sizeof(float));	
	hipMemcpy(d_a,a,ROWS*COLS*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_x,x1,(ROWS)*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc(&d_y,S_PARAM*ROWS*sizeof(float));	

	int bR       = NUM_THREADS-S_PARAM*2*HALF_BAND_SIZE;
	int num_blks = (ROWS + bR-1)/bR;



	hipFuncSetCacheConfig(reinterpret_cast<const void*>(matrix_powers_kernel_3), hipFuncCachePreferShared);
	cuda_sync();
	ts = timestamp();
        matrix_powers_kernel_3<<<num_blks,MATRIX_POWERS_NUM_THREADS>>>(d_a,d_x,d_y,m,bR,S_PARAM,0);
	cuda_sync();
	total = timestamp() - ts;
	hipMemcpy(y1,d_y,ROWS*S_PARAM*(sizeof(float)),hipMemcpyDeviceToHost);
	flops = S_PARAM*(2*num_entries - 1);
	gflops3 = flops / 1000000000.0;
	gflops3 = gflops3 / total;
	for(int i=0;i<m;i++)
	{
		if(y1[(S_PARAM-1)*ROWS+i] != y_host[i])
		{
		        Errors++;
	//		cout<<"Error at %d\n"<<i<<endl;
		}
		else
		{
	//		cout<<"SpMV output = "<<y_host[i]<<" Matrix Powers output = "<< y1[(S_PARAM-1)*ROWS+i]<<endl;
		}
  	}
	if(DISPLAY) 	
        	cout << "Matrix Powers: Reg-Blocking    " << total/flops << " seconds. " << gflops3 << " GFLOPS/s" <<"(m,band)"<<m<<","<<BAND_SIZE<< "  Errors = "<<Errors<<endl; 
	else
	       	cout << m<<"\t"<<S_PARAM<<"\t"<< gflops3 <<endl; 



	hipFree(d_a);
	hipFree(d_x);	
	hipFree(d_y);	
	
	delete[]a;	
	delete[]x1;	
	delete[]y1;	
      }

	
   }
}

