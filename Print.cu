
#include<cstdio>
#include"Print.h"
void PrintDeviceMatrix(float* A, int M, int N)
{
	float * A_1 = new float[M*N];
	float * A_2 = new float[M*N];
	
	
	hipblasGetMatrix(M,N,sizeof(float),A,M,A_1,M);
	FortrantoCFormat(M,N,A_1,A_2);
	PrintMatrix(A_2,M,N);

	delete [] A_1;
	delete [] A_2;

	
}
void PrintDeviceVector(float* A,  int N)
{
	float * A_1 = new float[N];
	
	
	hipblasGetVector(N,sizeof(float),A,1,A_1,1);
	PrintVector(A_1,N);

	delete [] A_1;

	
}
void PrintMatrix(double** A, int M, int N)
{
	printf("\n");
	for(int i =0;i < M;i++)
	{
		for(int j = 0;j<N;j++)
			printf(" %e\t",A[i][j]);
		printf("\n");
	}
}

void PrintMatrix(double* A, int M, int N)
{
	printf("\n");
	for(int i =0;i < M;i++)
	{	//printf("Row = %d\n",i);
		for(int j = 0;j<N;j++)
			printf(" %1.15f\t",A[j*M+i]);
		printf("\n");
	}
}

void PrintMatrix(float* A, int M, int N)
{
	printf("\n");
	for(int i =0;i < M;i++)
	{	//printf("Row = %d\n",i);
		for(int j = 0;j<N;j++)
			printf(" %1.15f\t",A[i*N+j]);
		printf("\n");
	}
}

void PrintVector(float* a, int n)
{
	for(int i =0;i < n;i++)
	{
		printf(" %f\t",a[i]);
	}

	printf("\n\n");
}

void PrintVector(double* a, int n)
{
	for(int i =0;i < n;i++)
	{
		printf(" %e\t",a[i]);
	}

	printf("\n\n");
}

void PrintVector(int* a, int n)
{
	for(int i =0;i < n;i++)
	{
		printf(" %d\t",a[i]);
	}

	printf("\n");
}
void CtoFortranFormat(int  M, int N, float* A, float* B)
{
	for(int i =0;i<N;i++)
	{
		for(int j =0;j<M;j++)
		{
			B[i*M+j] = A[j*N+i];
		}
	}
}


void FortrantoCFormat(int  M, int N, float* A, float* B)
{
	for(int i =0;i<M;i++)
	{
		for(int j =0;j<N;j++)
		{
			B[i*N+j] = A[j*M+i];
		}
	}
}


